#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <iostream>
#include <limits>
#include <sstream>
#include <string>
#include <thread>
#include <vector>

#define sleep(t) std::this_thread::sleep_for(std::chrono::milliseconds(t))
#define read_and_shift(arg_str, cnt) \
    std::string arg_str;             \
    if (cnt < argc) arg_str = argv[cnt++]

const float bytes_per_gb = (1 << 30);
const float ms_per_hour = 1000 * 3600;
const int max_grid_dim = (1 << 15);
const int max_block_dim = 1024;
const int max_sleep_time = 1e3;
const float sleep_interval = 1e16;
const int max_gpu_num = 32;

__global__ void default_script_kernel(char* array, size_t occupy_size) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= occupy_size) return;
    array[i]++;
}

void launch_default_script(char** array, size_t occupy_size,
                           std::vector<int>& grid_dim,
                           std::vector<int>& gpu_ids) {
    int gd = std::min(grid_dim[rand() % grid_dim.size()],
                      int(occupy_size / max_block_dim));
    for (int id : gpu_ids) {
        hipSetDevice(id);
        default_script_kernel<<<gd, max_block_dim, 0, NULL>>>(array[id],
                                                              occupy_size);
    }
}

void run_default_script(char** array, size_t occupy_size, float total_time,
                        std::vector<int>& gpu_ids) {
    // printf("Running default script >>>>>>>>>>>>>>>>>>>>\n");
    srand(time(NULL));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;
    std::vector<int> grid_dim;
    for (int i = 1; i <= max_grid_dim; i <<= 1) {
        grid_dim.push_back(i);
    }
    hipEventRecord(start, 0);

    // std::time_t now = std::time(0);
    // tm* localtm = localtime(&now);
    // std::cout << "Occupied since local time: " << asctime(localtm) << std::endl;
    int cnt = 0;
    while (true) {
        launch_default_script(array, occupy_size, grid_dim, gpu_ids);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        if (total_time >= 0 && time / ms_per_hour > total_time) break;
        if (!((++cnt) % size_t(sleep_interval / occupy_size))) {
            cnt = 0;
            // printf("Occupied time: %.2f hours\n", time / ms_per_hour);
            int sleep_time = rand() % max_sleep_time + 1;
            sleep(sleep_time);
        }
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void process_args(int argc, char** argv, std::vector<int>& gpu_ids, size_t& occupy_size, float& total_time, std::string& script_path, int& mode) {
    int cnt = 13;

    int gpu_num;
    hipGetDeviceCount(&gpu_num);
    srand(time(NULL));

    std::string id_str = argv[cnt++];

    std::replace(id_str.begin(), id_str.end(), ',', ' ');
    std::stringstream ss;
    ss << id_str;

    int id;
    while (ss >> id) {
        gpu_ids.push_back(id);
    }
    if (gpu_ids.size() == 1 && gpu_ids[0] == -1) {
        gpu_ids[0] = 0;
        for (int i = 1; i < gpu_num; ++i) {
            gpu_ids.push_back(i);
        }
    }

    mode = 0;
    if (cnt < argc) {
        sscanf(argv[cnt++], "%d", &mode);
    }

    total_time = -1;
    if (cnt < argc) {
        sscanf(argv[cnt++], "%f", &total_time);
    }

    float occupy_mem = -1;
    if (cnt < argc) {
        sscanf(argv[cnt++], "%f", &occupy_mem);
        occupy_size = size_t(occupy_mem * bytes_per_gb);
    }

    float random_left = 1;
    if (cnt < argc) {
        sscanf(argv[cnt++], "%f", &random_left);
    }

    if (occupy_mem == -1) {
        hipSetDevice(gpu_ids[0]);
        size_t total_size, avail_size;
        hipMemGetInfo(&avail_size, &total_size);
        occupy_size = total_size - size_t(bytes_per_gb * random_left);
    }

    script_path = "";
    if (cnt < argc) {
        script_path = argv[cnt++];
    }
}

void allocate_mem(char** array, size_t occupy_size, std::vector<int>& gpu_ids, int mode) {
    std::vector<size_t> allocated(max_gpu_num, 0);
    while (true) {
        // printf("Try allocate GPU memory %d times >>>>>>>>>>>>>>>>>>>>\n", ++cnt);
        int num_allocated = 0;
        for (int id : gpu_ids) {
            if (allocated[id] != occupy_size) {
                hipSetDevice(id);
                size_t total_size, avail_size;
                hipMemGetInfo(&avail_size, &total_size);
                size_t target_size = 0;

                if (mode != 2) {  // fight and occupy/release
                    target_size = min(avail_size - size_t(bytes_per_gb * 0.1), occupy_size - allocated[id]);
                } else {
                    // peace mode
                    // first occupy: av > 5G
                    // consecutive: wait until all finish
                    if (allocated[id] == 0) {
                        if (avail_size > size_t(5 * bytes_per_gb)) {
                            target_size = min(avail_size - size_t(bytes_per_gb * 2), occupy_size);
                        } else
                            target_size = 0;
                    } else {
                        target_size = occupy_size - allocated[id];
                    }
                }

                hipError_t status = hipMalloc(&array[id], target_size);
                if (status == hipSuccess) {
                    allocated[id] += target_size;
                    hipMemGetInfo(&avail_size, &total_size);
                    if (target_size >= size_t(bytes_per_gb * 0.5)) {
                        printf(
                            "[RUNNING] GPU-%d: Successfully allocate %.2f GB GPU memory (%.2f GB "
                            "available)\n",
                            id, target_size / bytes_per_gb, avail_size / bytes_per_gb);
                    }
                }
                if (allocated[id] == occupy_size) {
                    num_allocated++;
                    hipMemGetInfo(&avail_size, &total_size);
                    printf(
                        "[DONE] GPU-%d: Successfully allocate %.2f GB GPU memory (%.2f GB "
                        "available)\n",
                        id, occupy_size / bytes_per_gb, avail_size / bytes_per_gb);
                }
            }
        }
        // run_default_script(array, occupy_size, 2e-4, gpu_ids);
        if (num_allocated == gpu_ids.size()) break;
    }
    sleep(500);
    // inform_email(gpu_ids);
    // printf("Successfully allocate memory on all GPUs!\n");
}

void run_custom_script(char** array, std::vector<int>& gpu_ids,
                       std::string script_path) {
    printf("Running custom script >>>>>>>>>>>>>>>>>>>>\n");
    for (int id : gpu_ids) {
        hipFree(array[id]);
    }
    std::string cmd = "sh " + script_path;
    std::system(cmd.c_str());
}

int main(int argc, char** argv) {
    size_t occupy_size;
    float total_time;
    std::vector<int> gpu_ids;
    std::string script_path;
    char* array[max_gpu_num];
    int mode;

    process_args(argc, argv, gpu_ids, occupy_size, total_time, script_path, mode);
    // mode = 0: fight and occupy
    // mode = 1: fight and exit
    // mode = 2: peace

    if (mode == 1) total_time = 0.0;

    allocate_mem(array, occupy_size, gpu_ids, mode);

    if (script_path == "") {
        run_default_script(array, occupy_size, total_time, gpu_ids);
        for (int id : gpu_ids) {
            hipFree(array[id]);
        }
    } else {
        run_custom_script(array, gpu_ids, script_path);
    }

    return 0;
}
